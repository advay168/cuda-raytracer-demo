#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "header.h"

__host__ __device__ float fast_sqrtf(float x) {
  unsigned i = *(unsigned *)&x;
  i = (i + 0x3f76cf62) >> 1;
  float y = *(float *)&i;
  y = 0.5f * (y + x / y);
  y = 0.5f * (y + x / y);
  y = 0.5f * (y + x / y);
  y = 0.5f * (y + x / y);
  y = 0.5f * (y + x / y);
  y = 0.5f * (y + x / y);
  y = 0.5f * (y + x / y);
  y = 0.5f * (y + x / y);
  return y;
}

__host__ __device__ float Q_rsqrt(float number) {
  long i;
  float x2, y;
  const float threehalfs = 1.5F;

  x2 = number * 0.5F;
  y = number;
  i = *(long *)&y;           // evil floating point bit level hacking
  i = 0x5f3759df - (i >> 1); // what the fuck?
  y = *(float *)&i;
  y = y * (threehalfs - (x2 * y * y)); // 1st iteration
  //	y  = y * ( threehalfs - ( x2 * y * y ) );   // 2nd iteration, this can
  // be removed

  return y;
}

__device__ Colour scale(Colour c, float x) {
  return {
    (uint8_t)(c.r * x),
    (uint8_t)(c.g * x),
    (uint8_t)(c.b * x),
    (uint8_t)(c.a),
  };
}

__device__ Colour add(Colour c, Colour x) {
  return {
    (uint8_t)(c.r * x.r),
    (uint8_t)(c.g * x.g),
    (uint8_t)(c.b * x.b),
    (uint8_t)(c.a),
  };
}

__device__ vec3 operator+(const vec3 &a, const vec3 &b) {
  return {a.x + b.x, a.y + b.y, a.z + b.z};
}

__device__ vec3 operator-(const vec3 &a, const vec3 &b) {
  return {a.x - b.x, a.y - b.y, a.z - b.z};
}

__device__ float operator*(const vec3 &a, const vec3 &b) {
  return a.x * b.x + a.y * b.y + a.z * b.z;
}

__device__ vec3 operator*(const vec3 &a, const int b) {
  return {a.x * b, a.y * b, a.z * b};
}

__device__ vec3 operator/(const vec3 &a, const int b) {
  return {a.x / b, a.y / b, a.z / b};
}

__device__ float magnitude(const vec3 &a) { return fast_sqrtf(a * a); }
__device__ vec3 normalise(const vec3 &a) { return a / magnitude(a); }

__device__ bool intersect(const sphere &s, const vec3 &O, const vec3 &D, float t,
                          vec3 &hit_position, vec3 &hit_normal) {
  vec3 oc = O - s.center;
  float a = D * D;
  float b = 2.0f * (oc * D);
  float c = oc * oc - s.radius * s.radius;
  float discriminant = b * b - 4 * a * c;

  if (discriminant < 0)
    return false;

  float sqrt_disc = fast_sqrtf(discriminant);
  t = (-b - sqrt_disc) / (2.0f * a);

  if (t < 0)
    return false;

  hit_position = O + D * t;
  hit_normal = normalise(hit_position - s.center);
  return true;
}

__device__ float max_2(float a, float b) { return (a < b) ? b : a; }

__global__ void render(Colour *framebuffer, int width, int height,
                       sphere *spheres, int count) {
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;
  if (x >= width || y >= height)
    return;

  int index = y * width + x;
  float u = (2.0f * x / width - 1.0f) * 2.0f;
  float v = (2.0f * y / height - 1.0f) * 2.0f;
  vec3 ray_origin = {0, 0, 0};
  vec3 ray_direction = {u, v, 1};
  ray_direction = normalise(ray_direction);

  framebuffer[index] = Colour{0, 0,0,255}; // Default black

  float t_min = 1e9;
  vec3 hit_position;
  vec3 hit_normal;
  int hit_index = -1;

  for (int i = 0; i < count; i++) {
    float t;
    vec3 position;
    vec3 normal;
    if (intersect(spheres[i], ray_origin, ray_direction, t, position, normal) &&
        t < t_min) {
      t_min = t;
      hit_position = position;
      hit_normal = normal;
      hit_index = i;
    }
  }

  if (hit_index != -1) {
    vec3 light_pos = {0, 0, -1};
    vec3 light_dir = normalise(light_pos - hit_position);
    float intensity = 10 / (M_PI * 4 * (light_dir * light_dir)) ;
    intensity *= max_2(hit_normal * light_dir, 0);
    Colour colour = spheres[hit_index].colour;

    framebuffer[index] = add(Colour{30, 30, 30, 255}, scale(colour, intensity));
  }
}

Colour *d_framebuffer;
sphere *d_sphere = nullptr;
int d_count = 0;

void init(int width, int height) {
  hipFree(d_framebuffer);
  hipMalloc(&d_framebuffer, width * height * sizeof(uint32_t));
}

void doRender(Colour *framebuffer, int width, int height, sphere *spheres,
              int count) {
  if (count > d_count) {
    hipFree(d_sphere);
    d_count = count;
    hipMalloc(&d_sphere, sizeof(sphere) * d_count);
  }
  hipMemcpy(d_sphere, spheres, count * sizeof(sphere), hipMemcpyHostToDevice);

  dim3 blocks(32, 32);
  dim3 threads(width / blocks.x, height / blocks.y);
  // printf("threads(%d, %d, %d)\n", threads.x, threads.y, threads.z);
  // printf("blocks(%d, %d, %d)\n", blocks.x, blocks.y, blocks.z);
  render<<<threads, blocks>>>(d_framebuffer, width, height, d_sphere, count);

  hipMemcpy(framebuffer, d_framebuffer, width * height * sizeof(uint32_t),
             hipMemcpyDeviceToHost);

  hipError_t errSync = hipGetLastError();
  hipError_t errAsync = hipDeviceSynchronize();
  if (errSync != hipSuccess)
    printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
  if (errAsync != hipSuccess)
    printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
}
